#include "hip/hip_runtime.h"
#include "../include/weinfer/model.h"
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float *a, const float *b, float *c, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

Model::Model(int size) : size(size), d_a(nullptr), d_b(nullptr), d_c(nullptr) {
    allocateMemory(size);
}

Model::~Model() {
    freeMemory();
}

void Model::allocateMemory(int size) {
    hipMalloc((void**)&d_a, size * sizeof(float));
    hipMalloc((void**)&d_b, size * sizeof(float));
    hipMalloc((void**)&d_c, size * sizeof(float));
}

void Model::freeMemory() {
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

void Model::addVectors(const float* a, const float* b, float* c) {
    hipMemcpy(d_a, a, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, size);

    hipMemcpy(c, d_c, size * sizeof(float), hipMemcpyDeviceToHost);
}
